#include "fractal.cuh"


int * d_image;


extern "C" bool
fractal_cuda_init(int width, int height)
{
    if (d_image != NULL) {
        return false;
    } else {
        return (hipMalloc(&d_image, sizeof(int) * width*height) == hipSuccess);
    }
}

extern "C" void
fractal_cuda_clean()
{
    hipFree(&d_image);
    d_image = NULL;
}
